#include "hip/hip_runtime.h"

/**
* Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

/**
* Matrix multiplication: C = A * B.
* Host code.
*
* This sample implements matrix multiplication as described in Chapter 3
* of the programming guide.
* It has been written for clarity of exposition to illustrate various CUDA
* programming principles, not with the goal of providing the most
* performant generic kernel for matrix multiplication.
*
* See also:
* V. Volkov and J. Demmel, "Benchmarking GPUs to tune dense linear algebra,"
* in Proc. 2008 ACM/IEEE Conf. on Superconducting (SC '08),
* Piscataway, NJ: IEEE Press, 2008, pp. Art. 31:1-11.
*/

// System includes
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#define WIN32
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define N_ITER 1

//CONSTANTS
#define MATRIX_MAX_SIZE 4096
#define MATRIX_MIN_SIZE 0
#define EPS 1.e-6


bool is_n_correct(int n);

/**
* Matrix multiplication (CUDA Kernel) on the device: C = A * B
*/
template <int BLOCK_SIZE> __global__ void
matrixMulCUDA(float *C, float *A, float *B, int n, int offset)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int aBegin = n * BLOCK_SIZE * by;
	int aEnd = aBegin + n;
	int aStep = BLOCK_SIZE;
	int bBegin = n * BLOCK_SIZE * bx;
	int bStep = BLOCK_SIZE;
	float Csub = 0;

	for (int a = aBegin, b = bBegin; a < aEnd; a += aStep, b += bStep)
	{
		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

		// Load the matrices from device memory
		// to shared memory; each thread loads
		// one element of each matrix
		As[ty][tx] = A[a + n * ty + tx];
		Bs[ty][tx] = B[b + n * ty + tx];

		__syncthreads();

		// Multiply the two matrices together;
		// each thread computes one element
		// of the block sub-matrix
#pragma unroll
		for (int k = 0; k < BLOCK_SIZE; ++k)
		{
			Csub += As[ty][k] * Bs[tx][k];
		}

		// Synchronize to make sure that the preceding
		// computation is done before loading two new
		// sub-matrices of A and B in the next iteration
		__syncthreads();
	}

	// Write the block sub-matrix to device memory;
	// each thread writes one element
	int c = n * BLOCK_SIZE * by + BLOCK_SIZE * bx;
	C[c + n * ty + tx] = Csub;
}

void constantInit(float *data, int size, float val)
{
	for (int i = 0; i < size; ++i)
		data[i] = val;
}

void randomInit(float * data, int size)
{
	for (int i = 0; i < size; ++i)
		data[i] = (float)rand() / RAND_MAX;
}

void htd_copy(const unsigned mat_mem_size, float* h,  float* d, hipStream_t stream)
{
	checkCudaErrors(hipMemcpyAsync(d, h, mat_mem_size, hipMemcpyHostToDevice, stream));
}

void dth_copy(const unsigned mat_mem_size, float* d_C, float* h_C, hipStream_t stream)
{
	checkCudaErrors(hipMemcpyAsync(h_C, d_C, mat_mem_size, hipMemcpyDeviceToHost, stream));
}

void transpose(float * mat, int n)
{
	for (int i = 0; i < n; i++)
		for (int j = i+1; j < n; j++)
		{
			float temp = mat[i*n + j];
			mat[i*n + j] = mat[j*n + i];
			mat[j*n + i] = temp;
		}
}
void ikj(float * a, float * b, float *c, int n) {
	for (int i = 0; i < n; i++) {
		for (int k = 0; k < n; k++) {
			for (int j = 0; j < n; j++) {
				c[i*n + j] += a[i*n +k] * b[k*n +j];
			}
		}
	}
}
/**
* Run a simple test of matrix multiplication using CUDA
*/
int matrixMultiply(const int block_size, const int n, const int nstreams)
{
	if (nstreams < 1 || nstreams > n)
	{
		printf("Number of nstreams should be in the range [1, %d] in.\n", n);
		exit(0);
	}
	if (n % nstreams != 0)
	{
		printf("N should be a multiple of the number of nstreams.");
		exit(0);
	}
	// Allocate host memory for matrices A and B
	checkCudaErrors(hipSetDeviceFlags(hipDeviceScheduleBlockingSync | hipDeviceMapHost));
	
	const unsigned int mat_size_in_1d = n*n;
	const unsigned int mat_mem_size = sizeof(float) * mat_size_in_1d;
	float *h_a, *h_b, *h_c;
	checkCudaErrors(hipHostMalloc(&h_a, mat_mem_size));
	checkCudaErrors(hipHostMalloc(&h_b, mat_mem_size));
	checkCudaErrors(hipHostMalloc(&h_c, mat_mem_size));

	randomInit(h_a, mat_size_in_1d);
	randomInit(h_b, mat_size_in_1d);
	transpose(h_b, n);
	// Allocate device memory
	float *d_A, *d_B, *d_C;
	
	//nstreams for async communication
	hipStream_t *streams = static_cast<hipStream_t *>(malloc(nstreams * sizeof(hipStream_t)));
	for (int i = 0; i < nstreams; i++)
	{
		checkCudaErrors(hipStreamCreate(&streams[i]));
	}

	if (h_c == NULL)
	{
		fprintf(stderr, "Failed to allocate host matrix C!\n");
		exit(EXIT_FAILURE);
	}

	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_A), mat_mem_size));
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B), mat_mem_size));
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_C), mat_mem_size));

	// Setup execution parameters
	
	
	// Allocate CUDA events that we'll use for timing
	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreateWithFlags(&start, hipEventBlockingSync));
	checkCudaErrors(hipEventCreateWithFlags(&stop, hipEventBlockingSync));

	// Record the start event
	checkCudaErrors(hipEventRecord(start, NULL));
	
	const int step = mat_size_in_1d / nstreams;
	const unsigned int chunk_mem_size = mat_mem_size / nstreams;
	dim3 threads;
	if (nstreams > 1)
	{
		threads = dim3(1, 1);
	}
	else
	{
		threads = dim3(block_size, block_size);
	}
	const int grid_size = n / block_size;
	dim3 grid(grid_size, grid_size);

	for (int j = 0; j < N_ITER; j++)
	{
		printf("iteration %d\n", j);
		
		for (int i = 0, off = 0; i < nstreams; i++, off += step)
		{
			float* h_a_step = h_a + off; float* d_a_step = d_A + off;
			htd_copy(chunk_mem_size, h_a_step, d_a_step, streams[i]);

			float* h_b_step = h_b + off; float* d_b_step = d_B + off;
			htd_copy(chunk_mem_size, h_b_step, d_b_step, streams[i]);

			float* d_c_step = d_C + off; float* h_c_step = h_c + off;
			switch (block_size)
			{
			case 8: matrixMulCUDA<8> <<< grid, threads, 0, streams[i] >>> (d_c_step, d_a_step, d_b_step, n, off); break;
			case 16: matrixMulCUDA<16> <<< grid, threads, 0, streams[i] >>> (d_c_step, d_a_step, d_b_step, n, off); break;
			case 32: matrixMulCUDA<32> <<< grid, threads, 0, streams[i] >>> (d_c_step, d_a_step, d_b_step, n, off); break;
			}
			dth_copy(chunk_mem_size, d_c_step, h_c_step, streams[i]);
		}
		hipDeviceSynchronize();
	}

	// Record the stop event
	checkCudaErrors(hipEventRecord(stop, NULL));

	// Wait for the stop event to complete
	checkCudaErrors(hipEventSynchronize(stop));

	float msecTotal = 0.0f;
	checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
	// Compute and print the performance
	float msecPerMatrixMul = msecTotal / N_ITER;
	double flopsPerMatrixMul = 2.0 * (double)n * (double)n * (double)n;
	double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
	printf(
		"Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops, WorkgroupSize= %u threads/block\n",
		gigaFlops,
		msecPerMatrixMul,
		flopsPerMatrixMul,
		threads.x * threads.y);

	printf("Checking computed result for correctness: ");
	bool correct = true;
	float *cres = static_cast<float*>(malloc(mat_mem_size));
	constantInit(cres, mat_size_in_1d, 0);
	transpose(h_b, n);
	ikj(h_a, h_b, cres, n);
	float sum_org = 0, sum_cpy = 0;
	for (int i = 0; i < mat_size_in_1d; i++)
	{
		sum_org += h_c[i];
		sum_cpy += cres[i];
		const double abs_err = fabs(h_c[i] - cres[i]);
		const double abs_val = fabs(h_c[i]);
		const double rel_err = abs_err / abs_val / n;
		if (rel_err > EPS) {
			printf("Error - too big inaccuracy! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
				i, h_c[i], cres[i], EPS);
			correct = false;
		}
	}

	printf("%s\n", correct ? "OK" : "FAIL");
	printf("org- %f, cpy- %f, dif: %f \n", sum_org, sum_cpy, sum_org - sum_cpy);

	// Clean up memory
	free(h_a);
	free(h_b);
	free(h_c);
	free(streams);
	free(cres);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipDeviceReset();
	return correct ? EXIT_SUCCESS : EXIT_FAILURE;
}

bool is_n_correct(int n)
{
	return (n >= MATRIX_MIN_SIZE && n <= MATRIX_MAX_SIZE);
}

int get_n(int argc, char **argv, int block_size)
{
	int n = 512;
	if (checkCmdLineFlag(argc, (const char **)argv, "n"))
	{
		n = getCmdLineArgumentInt(argc, (const char **)argv, "n");
	}
	if (!is_n_correct(n))
	{
		printf("N=%d is incorrect. n should be in the range [%d, %d].\n", n, MATRIX_MIN_SIZE, MATRIX_MAX_SIZE);
		exit(-1);
	}
	if (n % block_size != 0)
	{
		printf("n should be multiplication of %d", block_size);
		exit(0);
	}
	return n;
}
/**
* Program main
*/
int main(int argc, char **argv)
{
	printf("[Matrix Multiply Using CUDA] - Starting...\n");
	// By default, we use device 0, otherwise we override the device ID based on what is provided at the command line
	int devID = 0;
	hipError_t error = hipGetDevice(&devID);

	if (error != hipSuccess)
	{
		printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
	}

	hipDeviceProp_t device_prop;
	error = hipGetDeviceProperties(&device_prop, devID);

	if (device_prop.computeMode == hipComputeModeProhibited)
	{
		fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
		exit(EXIT_SUCCESS);
	}

	if (error != hipSuccess)
	{
		printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
	}
	else
	{
		printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, device_prop.name, device_prop.major, device_prop.minor);
	}

	// Use a larger block size for Fermi and above
	const int block_size = (device_prop.major < 2) ? 16 : 32;

	// width of Matrix A
	const int n = get_n(argc, argv, block_size);
	printf("Matrix(%d,%d)\n", n, n);
	if (checkCmdLineFlag(argc, const_cast<const char **>(argv), "a")) //async
	{
		const int streams = 4;
		matrixMultiply(block_size, n, streams);
	} else
	{
		matrixMultiply(block_size, n, 1);
	}
	exit(0);
}
