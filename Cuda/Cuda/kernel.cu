#include "hip/hip_runtime.h"

/**
* Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

/**
* Matrix multiplication: C = A * B.
* Host code.
*
* This sample implements matrix multiplication as described in Chapter 3
* of the programming guide.
* It has been written for clarity of exposition to illustrate various CUDA
* programming principles, not with the goal of providing the most
* performant generic kernel for matrix multiplication.
*
* See also:
* V. Volkov and J. Demmel, "Benchmarking GPUs to tune dense linear algebra,"
* in Proc. 2008 ACM/IEEE Conf. on Superconducting (SC '08),
* Piscataway, NJ: IEEE Press, 2008, pp. Art. 31:1-11.
*/

// System includes
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#define WIN32
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

//CONSTANTS
const int MATRIX_MAX_SIZE = 4096;
const int MATRIX_MIN_SIZE = 0;
const int MAX_STREAMS = 10;
const double eps = 1.e-4;  // machine zero


bool is_n_correct(int n);

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

/**
* Matrix multiplication (CUDA Kernel) on the device: C = A * B
* n is A's width and wB is B's width
*/
template <int BLOCK_SIZE> __global__ void
matrixMulCUDA(float *C, float *A, float *B, int n)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Index of the first sub-matrix of A processed by the block
	int aBegin = n * BLOCK_SIZE * by;

	// Index of the last sub-matrix of A processed by the block
	int aEnd = aBegin + n - 1;

	// Step size used to iterate through the sub-matrices of A
	int aStep = BLOCK_SIZE;

	// Index of the first sub-matrix of B processed by the block
	int bBegin = BLOCK_SIZE * bx;

	// Step size used to iterate through the sub-matrices of B
	int bStep = BLOCK_SIZE * n;

	// Csub is used to store the element of the block sub-matrix
	// that is computed by the thread
	float Csub = 0;

	// Loop over all the sub-matrices of A and B
	// required to compute the block sub-matrix
	for (int a = aBegin, b = bBegin;
		a <= aEnd;
		a += aStep, b += bStep)
	{

		// Declaration of the shared memory array As used to
		// store the sub-matrix of A
		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

		// Declaration of the shared memory array Bs used to
		// store the sub-matrix of B
		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

		// Load the matrices from device memory
		// to shared memory; each thread loads
		// one element of each matrix
		As[ty][tx] = A[a + n * ty + tx];
		Bs[ty][tx] = B[b + n * ty + tx];

		// Synchronize to make sure the matrices are loaded
		__syncthreads();

		// Multiply the two matrices together;
		// each thread computes one element
		// of the block sub-matrix
#pragma unroll

		for (int k = 0; k < BLOCK_SIZE; ++k)
		{
			Csub += As[ty][k] * Bs[k][tx];
		}

		// Synchronize to make sure that the preceding
		// computation is done before loading two new
		// sub-matrices of A and B in the next iteration
		__syncthreads();
	}

	// Write the block sub-matrix to device memory;
	// each thread writes one element
	int c = n * BLOCK_SIZE * by + BLOCK_SIZE * bx;
	C[c + n * ty + tx] = Csub;
}

void constantInit(float *data, int size, float val)
{
	for (int i = 0; i < size; ++i)
		data[i] = val;
}

void htd_sync_copy(const unsigned mat_mem_size, float* h_A, float* h_B, float* d_A, float* d_B)
{
	// copy host memory to device
	hipError_t cuda_last_operation_status = hipMemcpy(d_A, h_A, mat_mem_size, hipMemcpyHostToDevice);
	if (cuda_last_operation_status != hipSuccess)
	{
		printf("hipMemcpy (d_A,h_A) returned error %s, line(%d)\n", hipGetErrorString(cuda_last_operation_status), __LINE__);
		exit(EXIT_FAILURE);
	}

	// copy host memory to device
	cuda_last_operation_status = hipMemcpy(d_B, h_B, mat_mem_size, hipMemcpyHostToDevice);
	if (cuda_last_operation_status != hipSuccess)
	{
		
		printf("hipMemcpy (d_B,h_B) returned error %s, line(%d)\n", hipGetErrorString(cuda_last_operation_status), __LINE__);
		exit(EXIT_FAILURE);
	}
}

void dth_sync_result_copy(const unsigned mat_mem_size, float* d_C, float* h_C)
{
	// Copy result from device to host
	const hipError_t cuda_last_operation_status = hipMemcpy(h_C, d_C, mat_mem_size, hipMemcpyDeviceToHost);
	if (cuda_last_operation_status != hipSuccess)
	{
		printf("hipMemcpy (h_C,d_C) returned error %s, line(%d)\n", hipGetErrorString(cuda_last_operation_status), __LINE__);
		exit(EXIT_FAILURE);
	}
}

/**
* Run a simple test of matrix multiplication using CUDA
*/
int matrixMultiply(int block_size, dim3 &dim, bool async, int streams = 0)
{
	const int n = dim.x;
	if (async)
	{
		if (streams < 1 || streams > MAX_STREAMS)
		{
			printf("Number of streams should be in the range [1, %d] in the async mode.\n", MAX_STREAMS);
			exit(0);
		}
		if (n % streams != 0)
		{
			printf("N should be a multiple of the number of streams.");
			exit(0);
		}
	}
	// Allocate host memory for matrices A and B
	const unsigned int mat_size = n*n;
	const unsigned int mat_mem_size = sizeof(float) * mat_size;
	float *h_A = (float *)malloc(mat_mem_size);
	float *h_B = (float *)malloc(mat_mem_size);

	// Initialize host memory
	const float valB = 0.01f;
	constantInit(h_A, mat_size, 1.0f);
	constantInit(h_B, mat_size, valB);

	// Allocate device memory
	float *d_A, *d_B, *d_C;

	float *h_C = (float *)malloc(mat_mem_size);

	//streams for async communication
	hipStream_t *available_streams = (hipStream_t *)malloc(streams * sizeof(hipStream_t));
	if (async)
	{
		for (int i = 0; i < streams; i++)
		{
			hipStreamCreate(&available_streams[i]);
		}
	}

	if (h_C == NULL)
	{
		fprintf(stderr, "Failed to allocate host matrix C!\n");
		exit(EXIT_FAILURE);
	}

	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_A), mat_mem_size));
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B), mat_mem_size));
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_C), mat_mem_size));

	// Setup execution parameters
	dim3 threads(block_size, block_size);
	dim3 grid(dim.x / threads.x, dim.y / threads.y);


	// Allocate CUDA events that we'll use for timing
	hipEvent_t start;
	checkCudaErrors(hipEventCreate(&start));
	hipEvent_t stop;
	checkCudaErrors(hipEventCreate(&stop));

	// Record the start event
	checkCudaErrors(hipEventRecord(start, NULL));
	
	int nIter = 10;
	for (int j = 0; j < nIter; j++)
	{
		printf("iteration %d\n", j);
		if (async)
		{
			int step = n / streams;
			for (int i = 0, off = 0; i < streams; i++, off += step)
			{
				//TO DO - set appropriate offsets
				//hipMemcpyAsync(d_A+off, h_A+off, );
				//switch (block_size)
				//{
				//case 8: matrixMulCUDA<8> << < grid, threads, 0, stream3 >> > (d_C, d_A + off, d_B + off, n / streams); break;
				//case 16: matrixMulCUDA<16> << < grid, threads, 0, stream3 >> > (d_C, d_A, d_B, n / streams); break;
				//case 32: matrixMulCUDA<32> << < grid, threads, 0, stream3 >> > (d_C, d_A, d_B, dim.x); break;
				//}
			}
		}
		else
		{
			htd_sync_copy(mat_mem_size, h_A, h_B, d_A, d_B);
			switch (block_size)
			{
			case 8: matrixMulCUDA<8> <<< grid, threads >>> (d_C, d_A, d_B, dim.x); break;
			case 16: matrixMulCUDA<16> <<< grid, threads >>> (d_C, d_A, d_B, dim.x); break;
			case 32: matrixMulCUDA<32> <<< grid, threads >>> (d_C, d_A, d_B, dim.x); break;
			}
			dth_sync_result_copy(mat_mem_size, d_C, h_C);
		}
	}
	

	// Record the stop event
	checkCudaErrors(hipEventRecord(stop, NULL));

	// Wait for the stop event to complete
	checkCudaErrors(hipEventSynchronize(stop));

	float msecTotal = 0.0f;
	checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
	// Compute and print the performance
	float msecPerMatrixMul = msecTotal / nIter;
	double flopsPerMatrixMul = 2.0 * (double)dim.x * (double)dim.y * (double)dim.x;
	double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
	printf(
		"Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops, WorkgroupSize= %u threads/block\n",
		gigaFlops,
		msecPerMatrixMul,
		flopsPerMatrixMul,
		threads.x * threads.y);

	printf("Checking computed result for correctness: ");
	bool correct = true;
	const double dot_length = dim.x;
	for (int i = 0; i < (int)(dim.x * dim.y); i++)
	{
		const double abs_err = fabs(h_C[i] - (dim.x * valB));
		const double abs_val = fabs(h_C[i]);
		const double rel_err = abs_err / abs_val / dot_length;

		if (rel_err > eps) {
			printf("Error - too big inaccuracy! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
				i, h_C[i], dim.x * valB, eps);
			correct = false;
		}
	}

	printf("%s\n", correct ? "OK" : "FAIL");

	// Clean up memory
	free(h_A);
	free(h_B);
	free(h_C);
	free(available_streams);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipDeviceReset();
	return correct ? EXIT_SUCCESS : EXIT_FAILURE;
}

bool is_n_correct(int n)
{
	return (n >= MATRIX_MIN_SIZE && n <= MATRIX_MAX_SIZE);
}

int get_n(int argc, char **argv, int block_size)
{
	int n = 512;
	if (checkCmdLineFlag(argc, (const char **)argv, "n"))
	{
		n = getCmdLineArgumentInt(argc, (const char **)argv, "n");
	}
	if (!is_n_correct(n))
	{
		printf("N=%d is incorrect. n should be in the range [%d, %d].\n", n, MATRIX_MIN_SIZE, MATRIX_MAX_SIZE);
		exit(-1);
	}
	if (n % block_size != 0)
	{
		printf("n should be multiplication of %d", block_size);
		exit(0);
	}
	return n;
}
/**
* Program main
*/
int main(int argc, char **argv)
{
	printf("[Matrix Multiply Using CUDA] - Starting...\n");
	// By default, we use device 0, otherwise we override the device ID based on what is provided at the command line
	int devID = 0;
	hipError_t error = hipGetDevice(&devID);

	if (error != hipSuccess)
	{
		printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
	}

	hipDeviceProp_t device_prop;
	error = hipGetDeviceProperties(&device_prop, devID);

	if (device_prop.computeMode == hipComputeModeProhibited)
	{
		fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
		exit(EXIT_SUCCESS);
	}

	if (error != hipSuccess)
	{
		printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
	}
	else
	{
		printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, device_prop.name, device_prop.major, device_prop.minor);
	}

	// Use a larger block size for Fermi and above
	const int block_size = (device_prop.major < 2) ? 16 : 32;

	dim3 dim(20 * block_size, 20 * block_size, 1);
	// width of Matrix A
	const int n = get_n(argc, argv, block_size);
	dim.x = dim.y = n;
	printf("Matrix(%d,%d)\n", dim.x, dim.y);
	if (checkCmdLineFlag(argc, (const char **) argv, "a")) //async
	{
		const int streams = (n / block_size);
		matrixMultiply(block_size, dim, true, streams);
	} else
	{
		matrixMultiply(block_size, dim, false);
	}
	exit(0);
}
