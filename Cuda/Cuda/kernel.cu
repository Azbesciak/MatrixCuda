#include "hip/hip_runtime.h"

/**
* Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

/**
* Matrix multiplication: C = A * B.
* Host code.
*
* This sample implements matrix multiplication as described in Chapter 3
* of the programming guide.
* It has been written for clarity of exposition to illustrate various CUDA
* programming principles, not with the goal of providing the most
* performant generic kernel for matrix multiplication.
*
* See also:
* V. Volkov and J. Demmel, "Benchmarking GPUs to tune dense linear algebra,"
* in Proc. 2008 ACM/IEEE Conf. on Superconducting (SC '08),
* Piscataway, NJ: IEEE Press, 2008, pp. Art. 31:1-11.
*/

// System includes
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define WIN32
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

/**
* Matrix multiplication (CUDA Kernel) on the device: C = A * B
* n is A's width and wB is B's width
*/
template <int BLOCK_SIZE> __global__ void
matrixMulCUDA(float *C, float *A, float *B, int n)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Index of the first sub-matrix of A processed by the block
	int aBegin = n * BLOCK_SIZE * by;

	// Index of the last sub-matrix of A processed by the block
	int aEnd = aBegin + n - 1;

	// Step size used to iterate through the sub-matrices of A
	int aStep = BLOCK_SIZE;

	// Index of the first sub-matrix of B processed by the block
	int bBegin = BLOCK_SIZE * bx;

	// Step size used to iterate through the sub-matrices of B
	int bStep = BLOCK_SIZE * n;

	// Csub is used to store the element of the block sub-matrix
	// that is computed by the thread
	float Csub = 0;

	// Loop over all the sub-matrices of A and B
	// required to compute the block sub-matrix
	for (int a = aBegin, b = bBegin;
		a <= aEnd;
		a += aStep, b += bStep)
	{

		// Declaration of the shared memory array As used to
		// store the sub-matrix of A
		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

		// Declaration of the shared memory array Bs used to
		// store the sub-matrix of B
		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

		// Load the matrices from device memory
		// to shared memory; each thread loads
		// one element of each matrix
		As[ty][tx] = A[a + n * ty + tx];
		Bs[ty][tx] = B[b + n * ty + tx];

		// Synchronize to make sure the matrices are loaded
		__syncthreads();

		// Multiply the two matrices together;
		// each thread computes one element
		// of the block sub-matrix
#pragma unroll

		for (int k = 0; k < BLOCK_SIZE; ++k)
		{
			Csub += As[ty][k] * Bs[k][tx];
		}

		// Synchronize to make sure that the preceding
		// computation is done before loading two new
		// sub-matrices of A and B in the next iteration
		__syncthreads();
	}

	// Write the block sub-matrix to device memory;
	// each thread writes one element
	int c = n * BLOCK_SIZE * by + BLOCK_SIZE * bx;
	C[c + n * ty + tx] = Csub;
}

void constantInit(float *data, int size, float val)
{
	for (int i = 0; i < size; ++i)
	{
		data[i] = val;
	}
}

/**
* Run a simple test of matrix multiplication using CUDA
*/
int matrixMultiply(int argc, char **argv, int block_size, dim3 &dim)
{
	// Allocate host memory for matrices A and B
	unsigned int size_A = dim.x * dim.y;
	unsigned int mat_mem_size = sizeof(float) * size_A;
	float *h_A = (float *)malloc(mat_mem_size);
	float *h_B = (float *)malloc(mat_mem_size);

	// Initialize host memory
	const float valB = 0.01f;
	constantInit(h_A, size_A, 1.0f);
	constantInit(h_B, size_A, valB);

	// Allocate device memory
	float *d_A, *d_B, *d_C;


	float *h_C = (float *)malloc(mat_mem_size);

	if (h_C == NULL)
	{
		fprintf(stderr, "Failed to allocate host matrix C!\n");
		exit(EXIT_FAILURE);
	}

	hipError_t error = hipMalloc((void **)&d_A, mat_mem_size);

	if (error != hipSuccess)
	{
		printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void **)&d_B, mat_mem_size);

	if (error != hipSuccess)
	{
		printf("hipMalloc d_B returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void **)&d_C, mat_mem_size);

	if (error != hipSuccess)
	{
		printf("hipMalloc d_C returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}

	// copy host memory to device
	error = hipMemcpy(d_A, h_A, mat_mem_size, hipMemcpyHostToDevice);

	if (error != hipSuccess)
	{
		printf("hipMemcpy (d_A,h_A) returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMemcpy(d_B, h_B, mat_mem_size, hipMemcpyHostToDevice);

	if (error != hipSuccess)
	{
		printf("hipMemcpy (d_B,h_B) returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}

	// Setup execution parameters
	dim3 threads(block_size, block_size);
	dim3 grid(dim.x / threads.x, dim.y / threads.y);

	// Create and start timer
	printf("Computing result using CUDA Kernel...\n");

	// Performs warmup operation using matrixMul CUDA kernel
	switch (block_size)
	{
	case 8: matrixMulCUDA<8> << < grid, threads >> > (d_C, d_A, d_B, dim.x); break;
	case 16: matrixMulCUDA<16> << < grid, threads >> > (d_C, d_A, d_B, dim.x); break;
	case 32: matrixMulCUDA<32> << < grid, threads >> > (d_C, d_A, d_B, dim.x); break;
	}
	printf("done\n");

	hipDeviceSynchronize();

	// Allocate CUDA events that we'll use for timing
	hipEvent_t start;
	error = hipEventCreate(&start);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	hipEvent_t stop;
	error = hipEventCreate(&stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Record the start event
	error = hipEventRecord(start, NULL);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Execute the kernel
	int nIter = 1;

	for (int j = 0; j < nIter; j++)
	{
		switch (block_size)
		{
		case 8: matrixMulCUDA<8> << < grid, threads >> > (d_C, d_A, d_B, dim.x); break;
		case 16: matrixMulCUDA<16> << < grid, threads >> > (d_C, d_A, d_B, dim.x); break;
		case 32: matrixMulCUDA<32> << < grid, threads >> > (d_C, d_A, d_B, dim.x); break;
		}
	}

	// Record the stop event
	error = hipEventRecord(stop, NULL);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Wait for the stop event to complete
	error = hipEventSynchronize(stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	float msecTotal = 0.0f;
	error = hipEventElapsedTime(&msecTotal, start, stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Compute and print the performance
	float msecPerMatrixMul = msecTotal / nIter;
	double flopsPerMatrixMul = 2.0 * (double)dim.x * (double)dim.y * (double)dim.x;
	double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
	printf(
		"Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops, WorkgroupSize= %u threads/block\n",
		gigaFlops,
		msecPerMatrixMul,
		flopsPerMatrixMul,
		threads.x * threads.y);

	// Copy result from device to host
	error = hipMemcpy(h_C, d_C, mat_mem_size, hipMemcpyDeviceToHost);

	if (error != hipSuccess)
	{
		printf("hipMemcpy (h_C,d_C) returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}

	printf("Checking computed result for correctness: ");
	bool correct = true;
	const double dot_length = dim.x;
	const double eps = 1.e-6;  // machine zero
	for (int i = 0; i < (int)(dim.x * dim.y); i++)
	{
		const double abs_err = fabs(h_C[i] - (dim.x * valB));
		const double abs_val = fabs(h_C[i]);
		const double rel_err = abs_err / abs_val / dot_length;

		if (rel_err > eps) {
			printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
				i, h_C[i], dim.x * valB, eps);
			correct = false;
		}
	}

	printf("%s\n", correct ? "OK" : "FAIL");

	// Clean up memory
	free(h_A);
	free(h_B);
	free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipDeviceReset();
	return correct ? EXIT_SUCCESS : EXIT_FAILURE;
}


/**
* Program main
*/
int main(int argc, char **argv)
{
	printf("[Matrix Multiply Using CUDA] - Starting...\n");
	// By default, we use device 0, otherwise we override the device ID based on what is provided at the command line
	int devID = 0;
	hipError_t error = hipGetDevice(&devID);

	if (error != hipSuccess)
	{
		printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
	}

	hipDeviceProp_t device_prop;
	error = hipGetDeviceProperties(&device_prop, devID);

	if (device_prop.computeMode == hipComputeModeProhibited)
	{
		fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
		exit(EXIT_SUCCESS);
	}

	if (error != hipSuccess)
	{
		printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
	}
	else
	{
		printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, device_prop.name, device_prop.major, device_prop.minor);
	}

	// Use a larger block size for Fermi and above
	const int block_size = (device_prop.major < 2) ? 16 : 32;

	dim3 dim(20 * block_size, 20 * block_size, 1);
	// width of Matrix A
	if (checkCmdLineFlag(argc, (const char **)argv, "n"))
	{
		int n = getCmdLineArgumentInt(argc, (const char **)argv, "n");
		dim.x = n;
		dim.y = n;
	}
	printf("Matrix(%d,%d)\n", dim.x, dim.y);

	const int matrix_result = matrixMultiply(argc, argv, block_size, dim);

	exit(matrix_result);
}
